#define kernel extern "C" __global__
#define device __device__
#define shared __shared__
#define syncThreads __syncthreads

#if __cplusplus > 199711L
#define HAS_CPP_11
#endif

#define BLOCK_SIZE 32

device int getGlobalID() {
    return blockIdx.x*blockDim.x + threadIdx.x;
}

kernel void position196 (/*__global*/ int *a) {
    const int i = getGlobalID();
    a[i] = 196;
}

kernel void positionBlockIdx(int *a) {
    const int i = getGlobalID();
    a[i] = blockIdx.x;
}

kernel void positionThreadIdx(int *a) {
    const int i = getGlobalID();
    a[i] = threadIdx.x;
}

kernel void positionGlobalIdx(int *a) {
    const int i = getGlobalID();
    a[i] = i;
}

//************************************************

kernel void raceCondition(/*__global*/ int* a) {
    *a += 50; //atomicAdd/Inc/Etc
}

//************************************************

__noinline__
__device__
int testNoInlineFunc(int* ptr) {
    int result = 0;
    for (int i = 0; i < ptr[0]; ++i)
        result += ptr[i];

    return result;

}

#ifdef HAS_CPP_11

#include <hip/hip_runtime.h>
#include "stdio.h"
struct A {
private:
    enum class Options {None, One, All};

    int a;
public:
    device A() {
        auto& b = a;
        if (&b != nullptr)
            b = [&]{ return 4; }();
        
        int arr[3] {0, 0, 0};
        for (auto& e: arr) {
            printf("%i\n", e);
        }
        
        static_assert(196 > 42, "This better compiles");
    }
    device ~A(){}
    device A(const A&&){}
    device virtual void foo() final { }
};
#endif //HAS_CPP_11

//************************************************

kernel void sum0(int* a, int* countPtr, int* result) {
    const int i = getGlobalID();
    
    const int count = *countPtr;
    
    if (i > count) {
        return;
    }
    atomicAdd(result, a[i]);
}

//************************************************

kernel void sum1(int* a, int* countPtr, int* result) {
    shared int partialSum;
    
    const int i = getGlobalID();
    const int count = *countPtr;
    
    if (i > count)
        return;
    
    if (threadIdx.x == 0)
        partialSum = 0;
    
    syncThreads();
    atomicAdd(&partialSum, a[i]);

    syncThreads();
    if (threadIdx.x == 0)
        atomicAdd(result, partialSum);
}

//************************************************

kernel void adjDiff0(int* result, int* input) {
    const int i = getGlobalID();
    
    if (i > 0) {
    
        int curr = input[i];
        int prev = input[i - 1];
        
        result[i] = curr - prev;
    }
}

//************************************************

kernel void adjDiff1(int* result, int* input) {
    int tx = threadIdx.x;
    
    shared int sharedData[BLOCK_SIZE]; //compile-time vs run-time
    
    const int i = getGlobalID();
    sharedData[tx] = input[i];
    //
    syncThreads();
    
    if (tx > 1)
        result[i] = sharedData[tx] - sharedData[tx - 1];
    else if (i > 1) {
        result[i] = sharedData[tx] - input[i - 1];
    }
}

//************************************************

kernel void badKernel0(int* foo) {
    shared int sharedInt;
    int* privatePtr = NULL;
    if (getGlobalID()%2) {
        privatePtr = &sharedInt;
    } else {
        privatePtr = foo;
    }
}

//************************************************

kernel void badKernel1(int* foo) { //hard crash
    shared int sharedInt;
    int* privatePtr = NULL;
    if (getGlobalID()%2) {
        syncThreads();
    } else {
        privatePtr = foo;
    }
}

//************************************************

kernel void matMul0(float* a, float* b, float* ab, int* widthPtr)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y; //get_global_id(0)
    const int column = blockIdx.x * blockDim.x + threadIdx.x;//get_global_id(1)
    const int width = *widthPtr;
    float res = 0;
    
    for (int k = 0; k < width; ++k)
        res += a[row * width + k] * b[k * width + column];
    
    ab[row * width + column] = res;
}

//************************************************
/*in sync with main.cpp::main::mul1::TILE_WIDTH*/
#define TILE_WIDTH 8

kernel void matMul1(float* a, float* b, float* ab, int* widthPtr) {
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    shared float sA[TILE_WIDTH][TILE_WIDTH];
    shared float sB[TILE_WIDTH][TILE_WIDTH];
    
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    
    float res = 0;
    
    const int width = *widthPtr;
  
    for (int p = 0; p < width/TILE_WIDTH; ++p) {
        sA[ty][tx] = a[row*width + (p*TILE_WIDTH + tx)];
        sB[ty][tx] = b[(p*TILE_WIDTH + ty)*width + col];
        
        syncThreads();
        
        for (int k = 0; k < TILE_WIDTH; ++k)
            res += sA[ty][k] * sB[k][tx];
        
        syncThreads();
    }
    
    ab[row*width + col] = res;
}

//************************************************

kernel void badMemoryAccess(int* input, int* output) {
    const int i = getGlobalID();
    
    int a = input[i];
    
    int STRIDE = 2;
    
    int b = input[i * STRIDE];

    output[i] = a + b;
}

//************************************************
//reduce example
kernel void blockSum(int* input, int* results, size_t* nPtr) {
    
    size_t n = *nPtr;
    
    shared int sharedData[BLOCK_SIZE];
    const int i = getGlobalID();
    const int tx = threadIdx.x;
    //
    if (threadIdx.x == 0) {
        for (int i = 0; i < BLOCK_SIZE; ++i)
            sharedData[i] = 0;
    }
    
    syncThreads();
    
    int x = 0;
    
    if (i >= n)
        return;
    
    x = input[i];
    
    sharedData[tx] = x;
    syncThreads();
    
    
    for (int offset = blockDim.x / 2;
         offset > 0;
         offset >>= 1)
    {
        if (tx < offset) {
            sharedData[tx] += sharedData[tx + offset];
        }
        syncThreads();
    }
    if (threadIdx.x == 0) {
        results[blockIdx.x] = sharedData[0];
    }
}

//0 1 2 3 4
//0 1 3 6 10
//************************************************
//results local to each block
kernel void inclusiveScan(int* data) {
    shared int sdata[BLOCK_SIZE];
    const int i = getGlobalID();
    
    int sum = data[i];
    
    sdata[threadIdx.x] = sum;
    
    syncThreads();
   
    for (int o = 1; o < blockDim.x; o <<= 1) {
        if (threadIdx.x >= o)
            sum += sdata[threadIdx.x - o];
        
        syncThreads();
    
        sdata[threadIdx.x] = sum;
        
        syncThreads();
    }
    
    data[i] = sdata[threadIdx.x];
}
